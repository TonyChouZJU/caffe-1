#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
template <typename Dtype>
void TripletSamplingLayer<Dtype>::Forward_gpu(
	const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
//		std::cout << bottom[1]->count() << std::endl;
	//sampling strategy, generate top, record top0_map, top1_map, top2_map.
		Dtype* bottom_label = bottom[1]->mutable_cpu_data();
//		std::cout << "hello" << std::endl;
		//store the label->dataIndex map.
		map<int, vector<int> > label_data_map;
		int max_label = 0;
		for(int i = 0; i < bottom[0]->num(); i++) {
//			std::cout << "hello:" << static_cast<int>(bottom_label[i]) << std::endl;
			 const int label_value = static_cast<int>(bottom_label[i]);
			// std::cout << "hello2" << std::endl;
			 if(label_value > max_label)
			 		max_label = label_value;
			 if(label_data_map.count(label_value) > 0){
			 	label_data_map[label_value].push_back(i);
			 }else{
			 	vector<int> tmp;
			 	tmp.push_back(i);
			 	label_data_map[label_value] = tmp;
			 }
		}
		if(label_data_map.size() == 1)
		{
			std::cout << "label number is 1" << std::endl;
			top[0]->Reshape(0, bottom[0]->channels(), 1,1 );
			top[1]->Reshape(0, bottom[0]->channels(), 1,1 );
			top[2]->Reshape(0, bottom[0]->channels(), 1,1 );
		}
		else{
			Dtype* anchors = top[0]->mutable_cpu_data();
			Dtype* positives = top[1]->mutable_cpu_data();
			Dtype* negatives = top[2]->mutable_cpu_data();
			int channels = bottom[0]->channels();
			Dtype* top0_map_Dtype = top0_map.mutable_cpu_data();
			Dtype* top1_map_Dtype = top1_map.mutable_cpu_data();
			Dtype* top2_map_Dtype = top2_map.mutable_cpu_data();
			for(int i = 0 ; i < bottom[0]->num(); i++)
			{
//				std::cout << "hello" << std::endl;
				const int label_value = static_cast<int>(bottom_label[i]);
//				std::cout << "hello2" << std::endl;
				//find positive data.
				int positive_index = i;
				if(label_data_map[label_value].size() != 1)
				{
					while(positive_index == i)
						positive_index = label_data_map[label_value][rand() % label_data_map[label_value].size()];
				}
				//find negative data.
				int negative_label = label_value;
				while(negative_label == label_value || label_data_map.count(negative_label) == 0)
				{
					negative_label = rand() % (max_label + 1);
				}
				// std::cout << "hello2" << std::endl;
				int negative_index = label_data_map[negative_label][rand() % label_data_map[negative_label].size()];
				const Dtype* anchor = bottom[0]->cpu_data() + (i * channels); 
				caffe_copy(channels, anchor, anchors + (i * channels));
				const Dtype* positive = bottom[0]->cpu_data() + (positive_index * channels);
				caffe_copy(channels, positive, positives + (i * channels));
				const Dtype* negative = bottom[0]->cpu_data() + (negative_index * channels);
				caffe_copy(channels, negative, negatives + (i * channels));
				 //std::cout << "hello3" << std::endl;
				top0_map_Dtype[i] = i;
				top1_map_Dtype[i] = positive_index;
				top2_map_Dtype[i] = negative_index;
			}
			Dtype* image_count_dtype = image_count.mutable_cpu_data();
			for(int i = 0 ; i < bottom[0]->num(); i++)
			{
				image_count_dtype[static_cast<int>(top0_map_Dtype[i])]++;
				image_count_dtype[static_cast<int>(top1_map_Dtype[i])]++;
				image_count_dtype[static_cast<int>(top2_map_Dtype[i])]++;
			}
		}
}
/*
template <typenameDtype>
__global__ void CLLBackward(const int count, const int channels, const Dtype margin, const Dtype alpha, const Dtype* diff, const Dtype* dist_sq_ap_, const Dtype* dist_sq_an_, Dtype *bottom_diff)
{
	CUDA_KERNEL_LOOP(i, count) {
		int n = i/channels; 
		Dtype mdist(0.0);
		mdist = margin + dist_sq_ap_[n] - dist_sq_an_[n];
		if(mdist > 0.0){
			bottom_diff[i] = alpha * diff[i];
		}else{
			bottom_diff[i] = 0;
		}
	}
}
*/
template <typename Dtype>
void TripletSamplingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, 
	const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{
	int num = top[0]->num();
	int channels = top[0]->channels();
	//same image diff accumulate and average.
	if(propagate_down[0]){
		const Dtype* top0_map_Dtype = top0_map.mutable_cpu_data();
		const Dtype* top1_map_Dtype = top1_map.mutable_cpu_data();
		const Dtype* top2_map_Dtype = top2_map.mutable_cpu_data();
		Dtype* bout = bottom[0]->mutable_cpu_diff();
		const Dtype* image_count_dtype = image_count.mutable_cpu_data();
		for(int i = 0 ; i < num; ++i)
		{
			caffe_cpu_axpby(
						channels, 
						Dtype(1.0), 
						top[0]->cpu_diff() + (i * channels),
						Dtype(0.0),
						bout + (static_cast<int>(top0_map_Dtype[i])*channels)
						);
			caffe_cpu_axpby(
						channels, 
						Dtype(1.0), 
						top[1]->cpu_diff() + (i * channels),
						Dtype(0.0),
						bout + (static_cast<int>(top1_map_Dtype[i])*channels)
						);	
			caffe_cpu_axpby(
						channels, 
						Dtype(1.0), 
						top[2]->cpu_diff() + (i * channels),
						Dtype(0.0),
						bout + (static_cast<int>(top2_map_Dtype[i])*channels)
						);	
		}
	}
}
INSTANTIATE_LAYER_GPU_FUNCS(TripletSamplingLayer);
}//namespace caffe

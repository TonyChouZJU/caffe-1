#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {
template <typename Dtype>
void TripletLossLayer<Dtype>::Forward_gpu(
	const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
	const int count = bottom[0]->count();
	caffe_gpu_sub(
		count, 
		bottom[0]->gpu_data(), //a
		bottom[1]->gpu_data(), //p
		diff_ap_.mutable_gpu_data() //a - p
	);
	caffe_gpu_sub(
		count, 
		bottom[0]->gpu_data(), //a
		bottom[2]->gpu_data(), //n
		diff_an_.mutable_gpu_data() //a - n
	);
	caffe_gpu_sub(
		count, 
		bottom[1]->gpu_data(), //p
		bottom[2]->gpu_data(), //n
		diff_pn_.mutable_gpu_data() //p - n
	);
	//power every element
	caffe_gpu_powx(
		count,
		diff_ap_.mutable_gpu_data(),
		Dtype(2),
		diff_sq_ap_.mutable_gpu_data()  // (a-p)^2
	);
	//y = op(A)*x + belta * y
	caffe_gpu_gemv(
		CblasNoTrans,
		bottom[0]->num(),
		bottom[0]->channels(),
		Dtype(1.0), //alpha = 1
		diff_sq_ap_.gpu_data(),  // (a-p)^2
		summer_vec_.gpu_data(),
		Dtype(0.0),
		dist_sq_ap_.mutable_gpu_data()
	);
	caffe_gpu_powx(
		count,
		diff_an_.mutable_gpu_data(), // (a-n)
		Dtype(2),
		diff_sq_an_.mutable_gpu_data()  // (a-n)^2
	);
	caffe_gpu_gemv(
		CblasNoTrans,
		bottom[0]->num(),
		bottom[0]->channels(),
		Dtype(1.0), //alpha = 1
		diff_sq_an_.gpu_data(),  // (a-n)^2
		summer_vec_.gpu_data(),
		Dtype(0.0),
		dist_sq_an_.mutable_gpu_data()
	);

	Dtype margin = this->layer_param_.triplet_loss_param().margin();
	Dtype loss(0.0);
//	const Dtype* sampleW = bottom[3]->cpu_data();
	for(int i = 0 ; i < bottom[0]->num(); i++)
	{
		loss +=  std::max(margin + dist_sq_ap_.cpu_data()[i] - dist_sq_an_.cpu_data()[i], Dtype(0.0));
	}
	loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
	top[0]->mutable_cpu_data()[0] = loss;

}

template <typename Dtype>
__global__ void CLLBackward(const int count, const int channels, const Dtype margin, const Dtype alpha, const Dtype* diff, const Dtype* dist_sq_ap_, const Dtype* dist_sq_an_, Dtype *bottom_diff)
{
	CUDA_KERNEL_LOOP(i, count) {
		int n = i/channels; 
		Dtype mdist(0.0);
		mdist = margin + dist_sq_ap_[n] - dist_sq_an_[n];
		if(mdist > 0.0){
			bottom_diff[i] = alpha * diff[i];
		}else{
			bottom_diff[i] = 0;
		}
	}
}

template <typename Dtype>
void TripletLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, 
	const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{
	Dtype margin = this->layer_param_.triplet_loss_param().margin();
	const int count = bottom[0]->count();
	const int channels = bottom[0]->channels();

	for(int i = 0; i < 3; i++)
	{
		if(propagate_down[i]) {
			const Dtype sign = (i < 2) ? -1 : 1;
			const Dtype alpha = sign * top[0]->cpu_diff()[0] / static_cast<Dtype>(bottom[0]->num());
			if(i == 0) {
				//NOLINT_NEXT_LINE(whitespace/operators)
				CLLBackward<Dtype>
				<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>> (
				count, channels, margin, alpha,
				diff_pn_.gpu_data(),
				dist_sq_ap_.gpu_data(),
				dist_sq_an_.gpu_data(),
				bottom[i]->mutable_gpu_diff()
				);
				CUDA_POST_KERNEL_CHECK;
			}else if(i == 1){
				//NOLINT_NEXT_LINE(whitespace/operators)
				CLLBackward<Dtype>
				<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
				count, channels, margin, alpha, 
				diff_ap_.gpu_data(),
				dist_sq_ap_.gpu_data(),
				dist_sq_an_.gpu_data(),
				bottom[i]->mutable_gpu_diff()
				);
				CUDA_POST_KERNEL_CHECK;
			}else if(i == 2) {
				//NOLINT_NEXT_LINE(whitespace/operators)
				CLLBackward<Dtype>
				<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
				count, channels, margin, alpha,
				diff_an_.gpu_data(),
				dist_sq_ap_.gpu_data(),
				dist_sq_an_.gpu_data(),
				bottom[i]->mutable_gpu_diff()
				);
				CUDA_POST_KERNEL_CHECK;
			}

		}
	}
}
INSTANTIATE_LAYER_GPU_FUNCS(TripletLossLayer);
}//namespace caffe
